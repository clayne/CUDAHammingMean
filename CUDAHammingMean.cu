#include "hip/hip_runtime.h"
/*******************************************************************
*   CUDAHammingMean.cu
*   CUDAHammingMean
*
*	Author: Kareem Omar
*	kareem.omar@uah.edu
*	https://github.com/komrad36
*
*	Last updated Oct 8, 2016
*******************************************************************/
//
// Fastest GPU implementation of a brute-force
// Hamming-weight matrix for 512-bit binary descriptors.
//
// Yes, that means the DIFFERENCE in popcounts is used
// for thresholding, NOT the ratio. This is the CORRECT
// approach for binary descriptors.
//
// This laboriously crafted kernel is EXTREMELY fast.
// 43 BILLION comparisons per second on a stock GTX1080,
// enough to match nearly 38,000 descriptors per frame at 30 fps (!)
//
// A key insight responsible for much of the performance of
// this insanely fast CUDA kernel is due to
// Christopher Parker (https://github.com/csp256), to whom
// I am extremely grateful.
//
// CUDA CC 3.0 or higher is required.
//
// All functionality is contained in the files CUDAK2NN.h
// and CUDAK2NN.cu. 'main.cpp' is simply a sample test harness
// with example usage and performance testing.
//

#include "CUDAHammingMean.h"

__global__ void
#ifndef __INTELLISENSE__
__launch_bounds__(256, 0)
#endif
CUDAK2NN_kernel(const hipTextureObject_t tex_q, const int num_q, const uint64_t* __restrict__ g_training, const int num_t, uint32_t* const __restrict__ g_sums) {
	uint64_t train = *(g_training += threadIdx.x & 7);
	g_training += 8;
	uint64_t q[8];
	for (int i = 0, offset = ((threadIdx.x & 24) << 3) + (threadIdx.x & 7) + (blockIdx.x << 11) + (threadIdx.y << 8); i < 8; ++i, offset += 8) {
		const uint2 buf = tex1Dfetch<uint2>(tex_q, offset);
		asm("mov.b64 %0, {%1,%2};" : "=l"(q[i]) : "r"(buf.x), "r"(buf.y)); // some assembly required
	}
	uint32_t total = 0U;
#pragma unroll 6
	for (int t = 0; t < num_t; ++t, g_training += 8) {
		uint32_t dist[4];
		for (int i = 0; i < 4; ++i) dist[i] = __byte_perm(__popcll(q[i] ^ train), __popcll(q[i + 4] ^ train), 0x5410);
		for (int i = 0; i < 4; ++i) dist[i] += __shfl_xor(dist[i], 1);
		train = *g_training;
		if (threadIdx.x & 1) dist[0] = dist[1];
		if (threadIdx.x & 1) dist[2] = dist[3];
		dist[0] += __shfl_xor(dist[0], 2);
		dist[2] += __shfl_xor(dist[2], 2);
		if (threadIdx.x & 2) dist[0] = dist[2];
		dist[0] += __shfl_xor(dist[0], 4);
		total += __byte_perm(dist[0], 0U, threadIdx.x & 4 ? 0x5432U : 0x5410U);
	}
	const int idx = (blockIdx.x << 8) + (threadIdx.y << 5) + threadIdx.x;
	if (idx < num_q) g_sums[idx] = total;
}

void CUDAK2NN(const void* const __restrict d_t, const int num_t, const hipTextureObject_t tex_q, const int num_q, uint32_t* const __restrict d_sums) {
	CUDAK2NN_kernel<<<((num_q - 1) >> 8) + 1, { 32, 8 }>>>(tex_q, num_q, reinterpret_cast<const uint64_t*>(d_t), num_t, d_sums);
	hipDeviceSynchronize();
}
